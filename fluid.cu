#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string>
#include <fstream>
#include <iostream>
#include <sstream>
#include <iomanip>
#include <hip/hip_fp16.h>
#include "cutil_math.h"
#include "double_buffer.cpp"


// Container for simulation state
struct fluid_state {

    float3 impulseLoc;
    float impulseTemp;
    float impulseDensity;
    float impulseRadius;
    float f_weight;
    float cell_size;
    float time_step;
    int3 dim;
    int64_t nelems;
    int step;
    DoubleBuffer<float3> *velocity;
    DoubleBuffer<float> *density;
    DoubleBuffer<float> *temperature;
    DoubleBuffer<float> *pressure;
    float *diverge;

    fluid_state(int3 dims) {
        step = 0;
        dim = dims;
        nelems = dims.x*dims.y*dims.z;
        velocity = new DoubleBuffer<float3>(nelems);
        density = new DoubleBuffer<float>(nelems);
        temperature = new DoubleBuffer<float>(nelems);
        pressure = new DoubleBuffer<float>(nelems);
        hipMalloc( (void**) &diverge, sizeof(float)*nelems);
    }

    ~fluid_state() {
        delete velocity;
        delete density;
        delete temperature;
        delete pressure;
        hipFree(diverge);
    }
};


// A couple IO utility functions
std::string pad_number(int n)
{
    std::ostringstream ss;
    ss << std::setw( 7 ) << std::setfill( '0' ) << n;
    return ss.str();
}

void save_image(uint8_t *pixels, int3 img_dims, std::string name) {
    std::ofstream file(name, std::ofstream::binary);
    if (file.is_open()) {
        file << "P6\n" << img_dims.x << " " << img_dims.y << "\n" << "255\n";
        file.write((char *)pixels, img_dims.x*img_dims.y*3);
        file.close();
    } else {
        std::cout << "Could not open file :(\n";
    }
}

// GPU helper functions
inline __device__ int3 operator*(const dim3 a, const uint3 b) {
    return make_int3(a.x*b.x, a.y*b.y, a.z*b.z);
}

inline __device__ int3 operator+(dim3 a, int3 b) {
    return make_int3(a.x+b.x, a.y+b.y, a.z+b.z);
}

inline __device__ int get_voxel(int x, int y, int z, int3 d)
{
    return z*d.y*d.x + y*d.x + x;
}

template <typename T> inline __device__ T zero() { return 0.0; }

template <> inline __device__ float  zero<float>() { return 0.0f; }
template <> inline __device__ float3 zero<float3>() { return make_float3(0.0f); }

template <typename T>
inline __device__ T get_cell(int3 c, int3 d, T *vol) {
    if (c.x < 0 || c.y < 0 || c.z < 0 ||
        c.x >= d.x || c.y >= d.y || c.z >= d.z) {
        return zero<T>();
    } else {
        return vol[ get_voxel( c.x, c.y, c.z, d ) ];
    }
}

template <typename T>
inline __device__ T get_cellF(float3 p, int3 d, T *vol) {
    
    // bilinear interpolation
    float3 l = floor(p);
    int3 rp = make_int3(l);
    float3 dif = p-l;
    T sum = zero<T>();

    #pragma unroll
    for (int a=0; a<=1; a++) 
    {
        #pragma unroll
        for (int b=0; b<=1; b++)
        {
            #pragma unroll
            for (int c=0; c<=1; c++)
            {
                sum += abs(float(1-a)-dif.x) *
                       abs(float(1-b)-dif.y) *
                       abs(float(1-c)-dif.z) *
                    get_cell( make_int3( rp.x+a, rp.y+b, rp.z+c ), d, vol);
            }
        }
    }

    return sum;
}
   
// Convert single index into 3D coordinates
inline __device__ int3 mod_coords(int i, int d) {
    return make_int3( i%d, (i/d) % d, (i/(d*d)) );
}

template <typename T>
inline __device__ T read_shared(T *mem, dim3 c, 
    int3 blk_dim, int pad, int x, int y, int z)
{
    return mem[ get_voxel(c.x+pad+x, c.y+pad+y, c.z+pad+z, blk_dim) ];
}

template <typename T>
__device__ void load_shared(dim3 blkDim, dim3 blkIdx, 
    dim3 thrIdx, int3 vd, int sdim, T *shared, T *src) 
{
    int t_idx = thrIdx.z*blkDim.y*blkDim.x 
        + thrIdx.y*blkDim.x + thrIdx.x; 
    // Load sdim*sdim*sdim cube of memory into shared array 
    const int cutoff = (sdim*sdim*sdim)/2;
    if (t_idx < cutoff) {
        int3 sp = mod_coords(t_idx, sdim);
        sp = sp + blkDim*blkIdx - 1;
        shared[t_idx] = get_cell( sp, vd, src);
        sp = mod_coords(t_idx+cutoff, sdim);
        sp = sp + blkDim*blkIdx - 1;
        shared[t_idx+cutoff] = get_cell( sp, vd, src);
    }
}

// Simulation compute kernels
template <typename T>
__global__ void pressure_solve(T *div, T *p_src, T *p_dst, 
        int3 vd, float amount)
{
    __shared__ T loc[1024];
    const int padding = 1; // How far to load past end of cube
    const int sdim = blockDim.x+2*padding; // 10 with blockdim 8
    const int3 s_dims = make_int3(sdim, sdim, sdim);
    const int x = blockDim.x*blockIdx.x+threadIdx.x;
    const int y = blockDim.y*blockIdx.y+threadIdx.y;
    const int z = blockDim.z*blockIdx.z+threadIdx.z;

    load_shared(
        blockDim, blockIdx, threadIdx, vd, sdim, loc, p_src); 
    __syncthreads();

    if (x >= vd.x || y >= vd.y || z >= vd.z) return;
    
    T d = div[get_voxel(x,y,z, vd)];

    T p_sum = 
             read_shared(loc, threadIdx, s_dims, padding, -1,  0,  0);
    p_sum += read_shared(loc, threadIdx, s_dims, padding,  1,  0,  0);
    p_sum += read_shared(loc, threadIdx, s_dims, padding,  0, -1,  0);
    p_sum += read_shared(loc, threadIdx, s_dims, padding,  0,  1,  0);
    p_sum += read_shared(loc, threadIdx, s_dims, padding,  0,  0, -1);
    p_sum += read_shared(loc, threadIdx, s_dims, padding,  0,  0,  1);
    //avg /= 6.0;
    //avg -= o;

    p_dst[ get_voxel(x,y,z, vd) ] = (p_sum+amount*d)*0.166667;//o + avg*amount;
}

template <typename V, typename T>
__global__ void divergence(V *velocity, T *div, int3 vd, float half_cell)
{
    __shared__ V loc[1024];
    const int padding = 1; // How far to load past end of cube
    const int sdim = blockDim.x+2*padding; // 10 with blockdim 8
    const int3 s_dims = make_int3(sdim, sdim, sdim);
    const int x = blockDim.x*blockIdx.x+threadIdx.x;
    const int y = blockDim.y*blockIdx.y+threadIdx.y;
    const int z = blockDim.z*blockIdx.z+threadIdx.z;

    load_shared(
        blockDim, blockIdx, threadIdx, vd, sdim, loc, velocity); 
    __syncthreads();

    if (x >= vd.x || y >= vd.y || z >= vd.z) return;
    
    T d = 
         read_shared(loc, threadIdx, s_dims, padding,  1,  0,  0).x;
    d -= read_shared(loc, threadIdx, s_dims, padding, -1,  0,  0).x;
    d += read_shared(loc, threadIdx, s_dims, padding,  0,  1,  0).y;
    d -= read_shared(loc, threadIdx, s_dims, padding,  0, -1,  0).y;
    d += read_shared(loc, threadIdx, s_dims, padding,  0,  0,  1).z;
    d -= read_shared(loc, threadIdx, s_dims, padding,  0,  0, -1).z;
    d *= half_cell;

    div[ get_voxel(x,y,z, vd) ] = d;
}

template <typename V, typename T>
__global__ void subtract_pressure(V *v_src, V *v_dest, T *pressure, 
    int3 vd, float grad_scale)
{
    __shared__ T loc[1024];
    const int padding = 1; // How far to load past end of cube
    const int sdim = blockDim.x+2*padding; // 10 with blockdim 8
    const int3 s_dims = make_int3(sdim, sdim, sdim);
    const int x = blockDim.x*blockIdx.x+threadIdx.x;
    const int y = blockDim.y*blockIdx.y+threadIdx.y;
    const int z = blockDim.z*blockIdx.z+threadIdx.z;

    load_shared(
        blockDim, blockIdx, threadIdx, vd, sdim, loc, pressure); 
    __syncthreads();

    if (x >= vd.x || y >= vd.y || z >= vd.z) return;
    
    V old_v = get_cell(make_int3(x,y,z), vd, v_src);

    V grad;
    grad.x = 
        read_shared(loc, threadIdx, s_dims, padding,  1,  0,  0) - 
        read_shared(loc, threadIdx, s_dims, padding, -1,  0,  0);
    grad.y =
        read_shared(loc, threadIdx, s_dims, padding,  0,  1,  0) -
        read_shared(loc, threadIdx, s_dims, padding,  0, -1,  0);
    grad.z = 
        read_shared(loc, threadIdx, s_dims, padding,  0,  0,  1) -
        read_shared(loc, threadIdx, s_dims, padding,  0,  0, -1);

    v_dest[ get_voxel(x,y,z, vd) ] = old_v - grad*grad_scale;
}

template <typename V, typename T>
__global__ void advection( V *velocity, T *source, T *dest, int3 vd, 
    float time_step, float dissipation)
{
    const int x = blockDim.x*blockIdx.x+threadIdx.x;
    const int y = blockDim.y*blockIdx.y+threadIdx.y;
    const int z = blockDim.z*blockIdx.z+threadIdx.z;

    if (x >= vd.x || y >= vd.y || z >= vd.z) return;
    
    V vel = velocity[ get_voxel(x,y,z,vd) ];

    float3 np = make_float3(float(x),float(y),float(z)) - time_step*vel;
    
    dest[ get_voxel(x,y,z, vd) ] = dissipation * get_cellF(np, vd, source);
}

template <typename T>
__global__ void impulse( T *target, float3 c,
    float radius, T val, int3 vd)
{
    const int x = blockDim.x*blockIdx.x+threadIdx.x;
    const int y = blockDim.y*blockIdx.y+threadIdx.y;
    const int z = blockDim.z*blockIdx.z+threadIdx.z;

    if (x >= vd.x || y >= vd.y || z >= vd.z) return;
    
    float3 p = make_float3(float(x),float(y),float(z));
    
    float dist = length(p-c);

    if (dist < radius) {
        target[ get_voxel(x,y,z, vd) ] = val;
    }
}

template <typename T>
__global__ void soft_impulse( T *target, float3 c,
    float radius, T val, float speed, int3 vd)
{
    const int x = blockDim.x*blockIdx.x+threadIdx.x;
    const int y = blockDim.y*blockIdx.y+threadIdx.y;
    const int z = blockDim.z*blockIdx.z+threadIdx.z;

    if (x >= vd.x || y >= vd.y || z >= vd.z) return;
    
    float3 p = make_float3(float(x),float(y),float(z));
    
    float dist = length(p-c);

    T cur = target[ get_voxel(x,y,z, vd) ];

    if (dist < radius && cur < val) {
        target[ get_voxel(x,y,z, vd) ] = cur + speed*val;
    }
}

template <typename T>
__global__ void wavey_impulse( T *target, float3 c,
    float3 size, T base, float amp, float freq, int3 vd)
{
    const int x = blockDim.x*blockIdx.x+threadIdx.x;
    const int y = blockDim.y*blockIdx.y+threadIdx.y;
    const int z = blockDim.z*blockIdx.z+threadIdx.z;

    if (x >= vd.x || y >= vd.y || z >= vd.z) return;
    
    float3 p = make_float3(float(x),float(y),float(z));
    
    //float dist = length(p-c);
    float3 minC = c-size;
    float3 maxC = c+size;

    //T cur = target[ get_voxel(x,y,z, vd) ];

    if (p.x>minC.x && p.y>minC.y && p.z>minC.z &&
        p.x<maxC.x && p.y<maxC.y && p.z<maxC.z ) {
        float v = 0.5*(sin(freq*p.x)+sin(freq*p.z)+0.0);
        v = v*v*v*v*v;
        target[ get_voxel(x,y,z, vd) ] = base + amp*v;
    }
}


template <typename V, typename T>
__global__ void buoyancy( V *v_src, T *t_src, T *d_src, V *v_dest, 
    float amb_temp, float time_step, float buoy, float weight, int3 vd)
{
    const int x = blockDim.x*blockIdx.x+threadIdx.x;
    const int y = blockDim.y*blockIdx.y+threadIdx.y;
    const int z = blockDim.z*blockIdx.z+threadIdx.z;

    if (x >= vd.x || y >= vd.y || z >= vd.z) return;
   
    T temp = t_src[ get_voxel(x,y,z, vd)];
    V vel = v_src[ get_voxel(x,y,z, vd)];

    if (temp > amb_temp)
    {
        T dense = d_src[ get_voxel(x,y,z, vd)];
        vel.y += (time_step * (temp - amb_temp) * buoy - dense * weight);
    }
    
    v_dest[ get_voxel(x,y,z, vd)] = vel;
}

// Runs a single iteration of the simulation
void simulate_fluid( fluid_state& state)
{

    float measured_time=0.0f;
    hipEvent_t start, stop;
    hipEventCreate( &start );
    hipEventCreate( &stop  );

    const int s = 8;
    dim3 block( s, s, s );
    dim3 grid( (state.dim.x+s-1)/s, 
               (state.dim.y+s-1)/s, 
               (state.dim.z+s-1)/s );

    hipEventRecord( start, 0 );
    
    advection<<<grid,block>>>(
            state.velocity->readTarget(),
            state.velocity->readTarget(),
            state.velocity->writeTarget(),
            state.dim, state.time_step, 1.0);
    state.velocity->swap();

    advection<<<grid,block>>>(
            state.velocity->readTarget(),
            state.temperature->readTarget(),
            state.temperature->writeTarget(),
            state.dim, state.time_step, 0.998);
    state.temperature->swap();

    advection<<<grid,block>>>(
            state.velocity->readTarget(),
            state.density->readTarget(),
            state.density->writeTarget(),
            state.dim, state.time_step, 0.9999);
    state.density->swap();

    buoyancy<<<grid,block>>>( 
            state.velocity->readTarget(), 
            state.temperature->readTarget(),
            state.density->readTarget(),
            state.velocity->writeTarget(), 
            0.0f, state.time_step, 1.0f, state.f_weight, state.dim);
    state.velocity->swap();

    float3 location = state.impulseLoc;
    location.x += 75.0*sinf(-0.003f*float(state.step));
    location.y += 75.0*cosf(-0.003f*float(state.step));

    /*
    soft_impulse<<<grid,block>>>( 
            state.temperature->readTarget(), 
            location, state.impulseRadius, 
            state.impulseTemp, 0.01, state.dim);

    soft_impulse<<<grid,block>>>(
            state.density->readTarget(), 
            location, state.impulseRadius, 
            state.impulseDensity, 0.01, state.dim);
    */    

    divergence<<<grid,block>>>(
            state.velocity->readTarget(),
            state.diverge, state.dim, 0.5);

    // clear pressure
    impulse<<<grid,block>>>(
            state.pressure->readTarget(),
            make_float3(0.0), 1000000.0f,
            0.0f, state.dim);
    
    for (int i=0; i<35; i++)
    {
        pressure_solve<<<grid,block>>>( 
                state.diverge,
                state.pressure->readTarget(),
                state.pressure->writeTarget(), 
                state.dim, -1.0);
        state.pressure->swap();
    }

    subtract_pressure<<<grid,block>>>(
            state.velocity->readTarget(),
            state.velocity->writeTarget(),
            state.pressure->readTarget(), 
            state.dim, 1.0);
    state.velocity->swap();

    hipEventRecord( stop, 0 );
    hipDeviceSynchronize();
    hipEventElapsedTime( &measured_time, start, stop );

    hipEventDestroy( start );
    hipEventDestroy( stop );

    std::cout << "Simulation Time: " << measured_time << "\n";
}

__device__ float2 rotate(float2 p, float a)
{
    return make_float2(p.x*cos(a) - p.y*sin(a),
                       p.y*cos(a) + p.x*sin(a));
}

// GPU volumetric raymarcher
__global__ void render_pixel( uint8_t *image, float *volume, 
        float *temper, int3 img_dims, int3 vol_dims, float step_size, 
        float3 light_dir, float3 cam_pos, float rotation)
{
    const int x = blockDim.x*blockIdx.x+threadIdx.x;
    const int y = blockDim.y*blockIdx.y+threadIdx.y;
    if (x >= img_dims.x || y >= img_dims.y) return;

    int3 vd = make_int3(vol_dims.x, vol_dims.y, vol_dims.z);
    // Create Normalized UV image coordinates
    float uvx =  float(x)/float(img_dims.x)-0.5;
    float uvy = -float(y)/float(img_dims.y)+0.5;
    uvx *= float(img_dims.x)/float(img_dims.y);     

    float3 v_center = make_float3(
            0.5*float(vol_dims.x),
            0.5*float(vol_dims.y),
            0.5*float(vol_dims.z));

    // Set up ray originating from camera
    float3 ray_pos = cam_pos-v_center;
    float2 pos_rot = rotate(make_float2(ray_pos.x, ray_pos.z), rotation);
    ray_pos.x = pos_rot.x;
    ray_pos.z = pos_rot.y;
    ray_pos += v_center;
    float3 ray_dir = normalize(make_float3(uvx,uvy,0.5));
    float2 dir_rot = rotate(make_float2(ray_dir.x, ray_dir.z), rotation);
    ray_dir.x = dir_rot.x;
    ray_dir.z = dir_rot.y;
    const float3 dir_to_light = normalize(light_dir);
    const float occ_thresh = 0.001;
    float d_accum = 1.0;
    float light_accum = 0.0;
    float temp_accum = 0.0;

    // Trace ray through volume
    for (int step=0; step<512; step++) {
        // At each step, cast occlusion ray towards light source
        float c_density = get_cellF(ray_pos, vd, volume);
        float3 occ_pos = ray_pos;
        ray_pos += ray_dir*step_size;
        // Don't bother with occlusion ray if theres nothing there
        if (c_density < occ_thresh) continue;
        float transparency = 1.0;
        for (int occ=0; occ<512; occ++) {
            transparency *= fmax(1.0-get_cellF(occ_pos, vd, volume),0.0);
            if (transparency < occ_thresh) break;
            occ_pos += dir_to_light*step_size;
        }
        d_accum *= fmax(1.0-c_density,0.0);
        light_accum += d_accum*c_density*transparency;
        if (d_accum < occ_thresh) break;
    }

    // gamma correction
    light_accum = pow(light_accum, 0.45);
    const int pixel = 3*(y*img_dims.x+x);
    image[pixel+0] = (uint8_t)(fmin(255.0*light_accum, 255.0));
    image[pixel+1] = (uint8_t)(fmin(255.0*light_accum, 255.0));
    image[pixel+2] = (uint8_t)(fmin(255.0*light_accum, 255.0));
}

void render_fluid(uint8_t *render_target, int3 img_dims, 
    float *d_volume, float *temper, int3 vol_dims, 
    float step_size, float3 light_dir, float3 cam_pos, float rotation) {

    float measured_time=0.0f;
    hipEvent_t start, stop;
    hipEventCreate( &start );
    hipEventCreate( &stop  );

    dim3 block( 32, 32 );
    dim3 grid( (img_dims.x+32-1)/32, (img_dims.y+32-1)/32 );

    hipEventRecord( start, 0 );

    // Allocate device memory for image
    int img_bytes = 3*sizeof(uint8_t)*img_dims.x*img_dims.y;
    uint8_t *device_img;
        hipMalloc( (void**)&device_img, img_bytes );
    if( 0 == device_img )
    {
        printf("couldn't allocate GPU memory\n");
                return;
    }

    render_pixel<<<grid,block>>>( 
        device_img, d_volume, temper, img_dims, vol_dims, 
        step_size, light_dir, cam_pos, rotation);

    // Read image back
    hipMemcpy( render_target, device_img, img_bytes, hipMemcpyDeviceToHost );

    hipEventRecord( stop, 0 );
    hipDeviceSynchronize();
    hipEventElapsedTime( &measured_time, start, stop );

    hipEventDestroy( start );
    hipEventDestroy( stop );

    std::cout << "Render Time: " << measured_time << "\n";
    hipFree(device_img);
}

int main(int argc, char* args[])
{

    const int3 vol_d = make_int3(512,512,512);
    const int3 img_d = make_int3(1920,1080,0);

    float3 cam;
    cam.x = static_cast<float>(vol_d.x)*0.5;
    cam.y = static_cast<float>(vol_d.y)*0.5;
    cam.z = 0.0;
    float3 light;
    light.x =  0.1;
    light.y =  1.0;
    light.z = -0.5;

    uint8_t *img = new uint8_t[3*img_d.x*img_d.y];
   
    fluid_state state(vol_d);
    
    state.impulseLoc = make_float3(0.5*float(vol_d.x),
                                   0.5*float(vol_d.y)-170.0,
                                   0.5*float(vol_d.z));
    state.impulseTemp = 4.0;
    state.impulseDensity = 0.35;
    state.impulseRadius = 18.0;
    state.f_weight = 0.05;
    state.time_step = 0.1;

    dim3 full_grid(vol_d.x/8+1, vol_d.y/8+1, vol_d.z/8+1);
    dim3 full_block(8,8,8);

    // zero out buffers
    impulse<<<full_grid, full_block>>>( state.velocity->readTarget(), 
        make_float3(0.0), 100000.0f, make_float3(0.0), vol_d);

    impulse<<<full_grid, full_block>>>( state.temperature->readTarget(), 
        make_float3(0.0), 100000.0f, 0.0f, vol_d);

    impulse<<<full_grid, full_block>>>( state.density->readTarget(), 
        make_float3(0.0), 100000.0f, 0.0f, vol_d);

    wavey_impulse<<<full_grid, full_block>>>( state.density->readTarget(),
        state.impulseLoc + make_float3(0.0, 70.0, 0.0),
        make_float3(100.0, 15.0, 100.0), 0.25f, 0.0f, 1.0f, vol_d);

    wavey_impulse<<<full_grid, full_block>>>( state.temperature->readTarget(),
        state.impulseLoc + make_float3(0.0, 70.0, 0.0),
        make_float3(100.0, 15.0, 100.0), 0.0f, 3.0f, 0.15f, vol_d);

    for (int f=0; f<=3000; f++) {
        
        std::cout << "Step " << f+1 << "\n";
        
        light.x = 1.05*sinf(0.006*float(state.step));
        light.z = 1.05*cosf(0.006*float(state.step));
        render_fluid(
                img, img_d, 
                state.density->readTarget(), 
                state.temperature->readTarget(),
                vol_d, 1.0, light, cam, 0.0*float(state.step));

        save_image(img, img_d, "output/R" + pad_number(f+1) + ".ppm");
        for (int st=0; st<1; st++) {
            simulate_fluid(state);
            state.step++;
        }
    }

    delete[] img;

    printf("CUDA: %s\n", hipGetErrorString( hipGetLastError() ) );

    hipDeviceReset();

    return 0;
}
